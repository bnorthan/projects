
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void multiply(float a, float b, float *c)
{
	*c=a*b;
}

int main()
{
  float a, b, c;
  float *c_pointer;
  a=1.35;
  b=2.5;

  hipMalloc((void**)&c_pointer, sizeof(float));
  multiply<<<1,1>>>(a, b, c_pointer);
  hipMemcpy(&c, c_pointer, sizeof(float),hipMemcpyDeviceToHost);
/*** This is C!!! You manage your garbage on your own!  ***/    
  hipFree(c_pointer);
  printf("Result = %f\n",c);
}