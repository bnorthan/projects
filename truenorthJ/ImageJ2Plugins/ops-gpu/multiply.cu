
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void multiply(float a, float b, float *c)
{
	*c=a*b;
}

extern "C"
__global__ void multiply2(float *a, float *b, float *c)
        /*************** Kernel Code **************/
{
        c[0]= a[0] * b[0];
}

int main()
{
  float a, b, c;
  float *c_pointer;
  a=1.35;
  b=2.5;

  hipMalloc((void**)&c_pointer, sizeof(float));
  multiply<<<1,1>>>(a, b, c_pointer);
  hipMemcpy(&c, c_pointer, sizeof(float),hipMemcpyDeviceToHost);
/*** This is C!!! You manage your garbage on your own!  ***/    
  hipFree(c_pointer);
  printf("Result = %f\n",c);
}
